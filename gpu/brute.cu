#include "hip/hip_runtime.h"
#include "brute.h"

struct DataStruct {
    float x, y, z, d, w;
};

__global__ void histo_kernel(DataStruct *b1, DataStruct *b2, long size, float *dsum, float *wsum, 
float min, float max, int nbins, int maxbins, bool diag) {
    // Temp histogram is dynamically allocated
    extern __shared__ float shared[];
    float *tempd = (float*) &shared[0];
    float *tempw = (float*) &shared[maxbins];
    // Initialize histogram bins to 0
    tempd[threadIdx.x] = 0; 
    tempw[threadIdx.x] = 0;
    // Need to sync threads up to make sure we don't start accumulating data in an
    // uninitialized bin
    __syncthreads();

    unsigned long i = threadIdx.x + blockIdx.x * blockDim.x; 
    unsigned long offset = blockDim.x * gridDim.x;

    float spacing = (max-min)/nbins;

    // This is a thread branching condition, will need to sync threads after this loop
    while (i < size) {
        float xi = b1[i].x;
        float yi = b1[i].y;
        float zi = b1[i].z;
        float di = b1[i].d;
        float wi = b1[i].w;
        for(unsigned long j = 0; j < size; ++j) {
            float dx = xi - b2[j].x;
            float dy = yi - b2[j].y;
            float dz = zi - b2[j].z;

            float separation = std::sqrt(dx*dx+dy*dy+dz*dz);
            float wgt = wi*b2[j].w;
            int index;
            if(separation < min){
                index = 0;
            }
            else if(separation >= max) {
                index = nbins+1;
            }
            else {
                index = floor((separation-min)/spacing)+1;
            }
            if(diag && j <= i) wgt = 0;
            // Notice that this index is not the same as the thread index!
            atomicAdd(&tempd[index], wgt*di*b2[j].d);
            atomicAdd(&tempw[index], wgt);
        }
        i += offset;
    }

    __syncthreads();

    atomicAdd(&(dsum[threadIdx.x]), tempd[threadIdx.x]);
    atomicAdd(&(wsum[threadIdx.x]), tempw[threadIdx.x]);
}

void bruteGPU(std::vector<std::vector<double> > &columns, double min, double max, 
int nbins, std::vector<double> &xi, long chunksize) {

    long nrows = columns[0].size();
    int nremainder = nrows % chunksize;
    if (nremainder > 0) {
        int npad = chunksize - nremainder;
        for(int i = 0; i < npad; ++i){
            columns[0].push_back(0);
            columns[1].push_back(0);
            columns[2].push_back(0);
            columns[3].push_back(0);
            columns[4].push_back(0);
        }
        nrows = columns[0].size();
    }
    assert(nrows % chunksize == 0);

    int nchunks = nrows / chunksize;

    std::cout << "nchunks: " << nchunks << std::endl;
    std::cout << "chunksize: " << chunksize << std::endl;

    DataStruct *data = (DataStruct*) malloc(nrows * sizeof(DataStruct));

    for(int i = 0; i < nrows; ++i) {
        data[i].x = columns[0][i];
        data[i].y = columns[1][i];
        data[i].z = columns[2][i];
        data[i].d = columns[3][i];
        data[i].w = columns[4][i];
    }

    std::cout << "sizeof data: " << nrows*sizeof(DataStruct)/1024./1024. << " MB" << std::endl;

    // Look up device properties
    hipDeviceProp_t prop;
    HANDLE_ERROR( hipGetDeviceProperties( &prop, 0 ) );

    // Lookup warpsize
    int warpSize = prop.warpSize;
    std::cout << "warp size: " << warpSize << std::endl;

    // Calculate how many threads per block to use
    int maxThreadsPerBlock = prop.maxThreadsPerBlock;
    int nWarpsPerBlock = 8;
    int threadsPerBlock = nWarpsPerBlock*warpSize;
    assert(threadsPerBlock < maxThreadsPerBlock);
    std::cout << "threadsPerBlock (used/max): " << threadsPerBlock << "/" << maxThreadsPerBlock << std::endl;

    // Check memory requirmenets
    long maxSharedMemoryPerBlock = prop.sharedMemPerBlock;
    long sharedMemoryPerBlock = 2*threadsPerBlock*sizeof(float);
    assert(sharedMemoryPerBlock <=  maxSharedMemoryPerBlock);
    std::cout << "Shared memory per block (used/max): " << sharedMemoryPerBlock << "/" << maxSharedMemoryPerBlock << std::endl;
    
    // Determine number of blocks to use
    int limitBlocksDueToSMem = maxSharedMemoryPerBlock / sharedMemoryPerBlock;
    int limitBlocksDueToWarps = threadsPerBlock / warpSize;
    int blocksPerMP = std::min(limitBlocksDueToSMem, limitBlocksDueToWarps);

    std::cout << "Active thread blocks per MP: " << blocksPerMP << std::endl;
    int blocks = blocksPerMP*prop.multiProcessorCount;
    std::cout << "Num blocks: " << blocks << std::endl;
    std::cout << "Total shared memory (used/max): " << sharedMemoryPerBlock*blocks << "/" << maxSharedMemoryPerBlock*blocks << std::endl;

    int nhistbins = threadsPerBlock;
    
    assert(nhistbins >= nbins+2);

    float dsum[nhistbins];
    float wsum[nhistbins];

    std::vector<double> tempxi(nbins,0);
    std::vector<double> counts(nbins,0);

    // allocate memory on the GPU for the file's data
    DataStruct *dev_b1, *dev_b2;
    float *dev_dsum;
    float *dev_wsum;

    HANDLE_ERROR( hipMalloc( (void**)&dev_b1, chunksize * sizeof(DataStruct) ) ); 
    HANDLE_ERROR( hipMalloc( (void**)&dev_b2, chunksize * sizeof(DataStruct) ) ); 
    HANDLE_ERROR( hipMalloc( (void**)&dev_dsum, nhistbins * sizeof( float ) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_wsum, nhistbins * sizeof( float ) ) );

    hipEvent_t start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );

    long totalcounts = 0;
    double totalElapsedTime = 0;
    for(int ichunk = 0; ichunk < nchunks; ++ichunk) {
        
        HANDLE_ERROR( hipEventRecord( start, 0 ) );

        for(int jchunk = 0; jchunk <= ichunk; ++jchunk) {

            //std::cout << "Starting chunk (" << ichunk << "," << jchunk << ")..." << std::endl;

            HANDLE_ERROR( hipMemcpy( dev_b1, &data[ichunk*chunksize], 
                chunksize * sizeof(DataStruct), hipMemcpyHostToDevice ) );
            HANDLE_ERROR( hipMemcpy( dev_b2, &data[jchunk*chunksize], 
                chunksize * sizeof(DataStruct), hipMemcpyHostToDevice ) );
            HANDLE_ERROR( hipMemset( dev_dsum, 0, nhistbins * sizeof( float ) ) );
            HANDLE_ERROR( hipMemset( dev_wsum, 0, nhistbins * sizeof( float ) ) );
        
            histo_kernel<<<blocks, threadsPerBlock, sharedMemoryPerBlock>>>(dev_b1, dev_b2, 
                chunksize, dev_dsum, dev_wsum, min, max, nbins, nhistbins, ichunk == jchunk);

            HANDLE_ERROR( hipMemcpy( dsum, dev_dsum, nhistbins * sizeof( float ), hipMemcpyDeviceToHost ) );
            HANDLE_ERROR( hipMemcpy( wsum, dev_wsum, nhistbins * sizeof( float ), hipMemcpyDeviceToHost ) );

            long chunkcounts = 0;
            // Save results from chunk
            //std::cout << wsum[0] << " " << wsum[nbins+1] << std::endl;
            for(int i = 0; i < nhistbins; ++i) {
                chunkcounts += wsum[i];
                if (i <= nbins && i > 0) {
                    //std::cout << i-1 << " " << dsum[i] << std::endl;
                    tempxi[i-1] += dsum[i];
                    counts[i-1] += wsum[i];
                }
            }
            totalcounts += chunkcounts;

            //std::cout << "Chunk (" << ichunk << "," << jchunk << ") counts: " << chunkcounts << std::endl;

            hipDeviceSynchronize();

        }
        // get stop time, and display the timing results
        HANDLE_ERROR( hipEventRecord( stop, 0 ) ); 
        HANDLE_ERROR( hipEventSynchronize( stop ) ); 
        float elapsedTime;
        HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
        totalElapsedTime += elapsedTime;
        printf( "Time to generate (%d):  %3.1f ms\n", ichunk, elapsedTime );
    }

    std::cout << "Total elapsed time: " << totalElapsedTime << " ms" << std::endl;

    long usedcounts = 0;
    for(int i = 0; i < nbins; ++i) {
        usedcounts += counts[i];
        if(counts[i] > 0) tempxi[i] /= counts[i];
    }

    tempxi.swap(xi);

    std::cout << "used " << usedcounts << " of " << totalcounts << " pairs." << std::endl;

    // Free host and device memory
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );
    hipFree( dev_dsum ); 
    hipFree( dev_wsum );
    hipFree( dev_b1 ); 
    hipFree( dev_b2 ); 
    free(data);

}