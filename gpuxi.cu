#include "hip/hip_runtime.h"
// Created 08-Aug-2013 by Daniel Margala (University of California, Irvine) <dmargala@uci.edu>
// Correlation function estimator using GPU.

// Compile using:
// nvcc -m64 -arch=sm_20 -lboost_program_options -llikely -lcosmo gpuxi.cu -o gpuxi

#include "cosmo/cosmo.h"
#include "likely/likely.h"

#include "boost/program_options.hpp"
#include "boost/format.hpp"

#include <iostream>
#include <fstream>
#include <cmath>
#include <vector>

#include <thrust/version.h>

#include "/Users/daniel/source/gpu/cuda_by_example/common/book.h"

namespace po = boost::program_options;
namespace lk = likely;

struct DataStruct {
    float x, y, z, d, w;
};

__global__ void histo_kernel(DataStruct *b1, DataStruct *b2, long size, float *dsum, float *wsum, int nbins, bool diag) {
    // Temp histogram is dynamically allocated
    extern __shared__ float shared[];
    float *tempd = (float*) &shared[0];
    float *tempw = (float*) &shared[nbins];
    tempd[threadIdx.x] = 0; 
    tempw[threadIdx.x] = 0;
    __syncthreads();

    unsigned long i = threadIdx.x + blockIdx.x * blockDim.x; 

    //printf("%lu\n",i);

    int offset = blockDim.x * gridDim.x;

    float separation;
    while (i < size) {
        float xi = b1[i].x;
        float yi = b1[i].y;
        float zi = b1[i].z;
        float di = b1[i].d;
        float wi = b1[i].w;
        for(unsigned long j = 0; j < size; ++j) {
            float dx = xi - b2[j].x;
            float dy = yi - b2[j].y;
            float dz = zi - b2[j].z;

            separation = std::sqrt(dx*dx+dy*dy+dz*dz);
            int index;
            if(separation <= 0 || separation >= 200){
                index = 255;
            }
            else {
                index = (int) (separation);
            }
            float wgt = wi*b2[j].w;
            if(diag && j <= i) wgt = 0;
            atomicAdd(&tempd[index], wgt*di*b2[j].d);
            atomicAdd(&tempw[index], wgt);
        }
        i += offset;
    }

    __syncthreads();

    atomicAdd(&(dsum[threadIdx.x]), tempd[threadIdx.x]);
    atomicAdd(&(wsum[threadIdx.x]), tempw[threadIdx.x]);
}

void bruteGPU(std::vector<std::vector<double> > const &columns, lk::BinnedGrid const &grid, bool rmu,
double x1min, double x1max, double x2min, double x2max, std::vector<double> &xi) {

    int chunksize = 1000;
    long nrows = columns[0].size()/250;

    int nchunks = nrows / chunksize;

    std::cout << "nchunks: " << nchunks << std::endl;
    std::cout << "chunksize: " << chunksize << std::endl;

    DataStruct *data = (DataStruct*) malloc(nrows * sizeof(DataStruct));

    for(int i = 0; i < nrows; ++i) {
        data[i].x = columns[0][i];
        data[i].y = columns[1][i];
        data[i].z = columns[2][i];
        data[i].d = columns[3][i];
        data[i].w = columns[4][i];
    }

    std::cout << "sizeof data: " << nrows*sizeof(DataStruct)/1024./1024. << " MB" << std::endl;

    // Look up device properties
    hipDeviceProp_t prop;
    HANDLE_ERROR( hipGetDeviceProperties( &prop, 0 ) );
    int blocks = 2*prop.multiProcessorCount; 
    int threadsPerBlock = 256;

    std::cout << "num blocks: " << blocks << std::endl;
    std::cout << "threadsPerBlock: " << threadsPerBlock << std::endl;

    int nhistbins = threadsPerBlock;

    float dsum[nhistbins];
    float wsum[nhistbins];

    std::vector<double> tempxi(50,0);
    std::vector<double> counts(50,0);

    // allocate memory on the GPU for the file's data
    DataStruct *dev_b1, *dev_b2;
    float *dev_dsum;
    float *dev_wsum;

    HANDLE_ERROR( hipMalloc( (void**)&dev_b1, chunksize * sizeof(DataStruct) ) ); 
    HANDLE_ERROR( hipMalloc( (void**)&dev_b2, chunksize * sizeof(DataStruct) ) ); 
    HANDLE_ERROR( hipMalloc( (void**)&dev_dsum, nhistbins * sizeof( float ) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_wsum, nhistbins * sizeof( float ) ) );

    double totalcounts = 0;

    hipEvent_t start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );

    std::cout << "shared memory per block: " << 2*threadsPerBlock*sizeof(float)/1024. << " KB" << std::endl;

    double totalElapsedTime = 0;
    for(int ichunk = 0; ichunk < nchunks; ++ichunk) {
        
        HANDLE_ERROR( hipEventRecord( start, 0 ) );

        for(int jchunk = ichunk; jchunk < nchunks; ++jchunk) {

            HANDLE_ERROR( hipMemcpy( dev_b1, &data[ichunk*chunksize], chunksize * sizeof(DataStruct), hipMemcpyHostToDevice ) );
            HANDLE_ERROR( hipMemcpy( dev_b2, &data[jchunk*chunksize], chunksize * sizeof(DataStruct), hipMemcpyHostToDevice ) );
            HANDLE_ERROR( hipMemset( dev_dsum, 0, nhistbins * sizeof( float ) ) );
            HANDLE_ERROR( hipMemset( dev_wsum, 0, nhistbins * sizeof( float ) ) );
        
            histo_kernel<<<blocks,
                           threadsPerBlock,
                           2*threadsPerBlock*sizeof(float)>>>( dev_b1, dev_b2, chunksize, dev_dsum, dev_wsum, nhistbins, ichunk == jchunk);

            //std::cout << "size of dsum: " << sizeof(dsum) << " B" << std::endl;

            HANDLE_ERROR( hipMemcpy( dsum, dev_dsum, nhistbins * sizeof( float ), hipMemcpyDeviceToHost ) );
            HANDLE_ERROR( hipMemcpy( wsum, dev_wsum, nhistbins * sizeof( float ), hipMemcpyDeviceToHost ) );

            // Check results
            for(int i = 0; i < nhistbins; ++i) {
                //dsum[i] += wsum[i];
                //std::cout << "dsum[" << i << "] = " << wsum[i] << std::endl;
                totalcounts += wsum[i];
                if(i < 200) {
                    tempxi[i/4] += dsum[i];
                    counts[i/4] += wsum[i];
                }
            }

            hipDeviceSynchronize();

        }
        // get stop time, and display the timing results
        HANDLE_ERROR( hipEventRecord( stop, 0 ) ); 
        HANDLE_ERROR( hipEventSynchronize( stop ) ); 
        float elapsedTime;
        HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
        totalElapsedTime += elapsedTime;
        printf( "Time to generate (%d):  %3.1f ms\n", ichunk, elapsedTime );
    }

    std::cout << "Total elapsed time: " << totalElapsedTime << std::endl;

    for(int i = 0; i < 50; ++i) {
        if(counts[i] > 0) tempxi[i] /= counts[i];
    }

    tempxi.swap(xi);

    std::cout << "Total counts: " << totalcounts/1000. << " s" << std::endl;

    // Free host and device memory
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );
    hipFree( dev_dsum ); 
    hipFree( dev_wsum );
    hipFree( dev_b1 ); 
    hipFree( dev_b2 ); 
    //free( buffer );

    free(data);

}

int main(int argc, char **argv) {

    // Configure command-line option processing
    std::string infile,outfile,axis1,axis2;
    po::options_description cli("Correlation function estimator");
    cli.add_options()
        ("help,h", "Prints this info and exits.")
        ("verbose", "Prints additional information.")
        ("input,i", po::value<std::string>(&infile)->default_value(""),
            "Filename to read field samples from")
        ("output,o", po::value<std::string>(&outfile)->default_value("xi.dat"),
            "Filename to write correlation function to")
        ("axis1", po::value<std::string>(&axis1)->default_value("[0:200]*50"),
            "Axis-1 binning")
        ("axis2", po::value<std::string>(&axis2)->default_value("[0:200]*50"),
            "Axis-2 binning")
        ("rmu", "Use (r,mu) binning instead of (rP,rT) binning")
        ;

    // do the command line parsing now
    po::variables_map vm;
    try {
        po::store(po::parse_command_line(argc, argv, cli), vm);
        po::notify(vm);
    }
    catch(std::exception const &e) {
        std::cerr << "Unable to parse command line options: " << e.what() << std::endl;
        return -1;
    }
    if(vm.count("help")) {
        std::cout << cli << std::endl;
        return 1;
    }
    bool verbose(vm.count("verbose")),rmu(vm.count("rmu")),useCPU(vm.count("cpu"));

    // Read the input file
    if(0 == infile.length()) {
        std::cerr << "Missing infile parameter." << std::endl;
        return -2;
    }
    std::vector<std::vector<double> > columns(5);
    try {
        std::ifstream in(infile.c_str());
        lk::readVectors(in,columns);
        in.close();
    }
    catch(std::exception const &e) {
        std::cerr << "Error while reading " << infile << ": " << e.what() << std::endl;
        return -3;
    }
    if(verbose) {
        std::cout << "Read " << columns[0].size() << " rows from " << infile
            << std::endl;
    }

    // Generate the correlation function grid and run the estimator
    std::vector<double> xi;
    try {
        lk::AbsBinningCPtr bins1 = lk::createBinning(axis1), bins2 = lk::createBinning(axis2);
        double x1min(bins1->getBinLowEdge(0)), x1max(bins1->getBinHighEdge(bins1->getNBins()-1));
        double x2min(bins2->getBinLowEdge(0)), x2max(bins2->getBinHighEdge(bins2->getNBins()-1));
        lk::BinnedGrid grid(bins1,bins2);
        bruteGPU(columns,grid,rmu,x1min,x1max,x2min,x2max,xi);
    }
    catch(std::exception const &e) {
        std::cerr << "Error while running the estimator: " << e.what() << std::endl;
    }

    // Save the estimator results
    try {
        std::ofstream out(outfile.c_str());
        for(int index = 0; index < xi.size(); ++index) {
            out << index << ' ' << xi[index] << std::endl;
        }
        out.close();
    }
    catch(std::exception const &e) {
        std::cerr << "Error while saving results: " << e.what() << std::endl;
    }

    return 0;
}
