#include "hip/hip_runtime.h"
// Created 08-Aug-2013 by Daniel Margala (University of California, Irvine) <dmargala@uci.edu>
// Correlation function estimator using GPU.

// Compile using:
// nvcc -m64 -arch=sm_20 -lboost_program_options -llikely -lcosmo gpuxi.cu -o gpuxi

// Example usage:
// time -p ./gpuxi -i /Users/daniel/Cosmo/LyAlpha/cosmo/build/delta.dat --verbose

#include "cosmo/cosmo.h"
#include "likely/likely.h"

#include "boost/program_options.hpp"
#include "boost/format.hpp"

#include <iostream>
#include <fstream>
#include <cmath>
#include <vector>

#include <thrust/version.h>

#include "/Users/daniel/source/gpu/cuda_by_example/common/book.h"

namespace po = boost::program_options;
namespace lk = likely;

struct DataStruct {
    float x, y, z, d, w;
};

__global__ void histo_kernel(DataStruct *b1, DataStruct *b2, long size, float *dsum, float *wsum, 
float min, float max, int nbins, int maxbins, bool diag) {
    // Temp histogram is dynamically allocated
    extern __shared__ float shared[];
    float *tempd = (float*) &shared[0];
    float *tempw = (float*) &shared[maxbins];
    // Initialize histogram bins to 0
    tempd[threadIdx.x] = 0; 
    tempw[threadIdx.x] = 0;
    // Need to sync threads up to make sure we don't start accumulating data in an
    // uninitialized bin
    __syncthreads();

    unsigned long i = threadIdx.x + blockIdx.x * blockDim.x; 
    unsigned long offset = blockDim.x * gridDim.x;

    float spacing = (max-min)/nbins;

    // This is a thread branching condition, will need to sync threads after this loop
    while (i < size) {
        float xi = b1[i].x;
        float yi = b1[i].y;
        float zi = b1[i].z;
        float di = b1[i].d;
        float wi = b1[i].w;
        for(unsigned long j = 0; j < size; ++j) {
            float dx = xi - b2[j].x;
            float dy = yi - b2[j].y;
            float dz = zi - b2[j].z;

            float separation = std::sqrt(dx*dx+dy*dy+dz*dz);
            float wgt = wi*b2[j].w;
            int index;
            if(separation < min){
                index = 0;
            }
            else if(separation >= max) {
                index = nbins+1;
            }
            else {
                index = floor((separation-min)/spacing)+1;
            }
            if(diag && j <= i) wgt = 0;
            // Notice that this index is not the same as the thread index!
            atomicAdd(&tempd[index], wgt*di*b2[j].d);
            atomicAdd(&tempw[index], wgt);
        }
        i += offset;
    }

    __syncthreads();

    atomicAdd(&(dsum[threadIdx.x]), tempd[threadIdx.x]);
    atomicAdd(&(wsum[threadIdx.x]), tempw[threadIdx.x]);
}

void bruteGPU(std::vector<std::vector<double> > &columns, double min, double max, 
int nbins, std::vector<double> &xi, long chunksize) {

    long nrows = columns[0].size();
    int nremainder = nrows % chunksize;
    if (nremainder > 0) {
        int npad = chunksize - nremainder;
        for(int i = 0; i < npad; ++i){
            columns[0].push_back(0);
            columns[1].push_back(0);
            columns[2].push_back(0);
            columns[3].push_back(0);
            columns[4].push_back(0);
        }
        nrows = columns[0].size();
    }
    assert(nrows % chunksize == 0);

    int nchunks = nrows / chunksize;

    std::cout << "nchunks: " << nchunks << std::endl;
    std::cout << "chunksize: " << chunksize << std::endl;

    DataStruct *data = (DataStruct*) malloc(nrows * sizeof(DataStruct));

    for(int i = 0; i < nrows; ++i) {
        data[i].x = columns[0][i];
        data[i].y = columns[1][i];
        data[i].z = columns[2][i];
        data[i].d = columns[3][i];
        data[i].w = columns[4][i];
    }

    std::cout << "sizeof data: " << nrows*sizeof(DataStruct)/1024./1024. << " MB" << std::endl;

    // Look up device properties
    hipDeviceProp_t prop;
    HANDLE_ERROR( hipGetDeviceProperties( &prop, 0 ) );

    // Lookup warpsize
    int warpSize = prop.warpSize;
    std::cout << "warp size: " << warpSize << std::endl;

    // Calculate how many threads per block to use
    int maxThreadsPerBlock = prop.maxThreadsPerBlock;
    int nWarpsPerBlock = 8;
    int threadsPerBlock = nWarpsPerBlock*warpSize;
    assert(threadsPerBlock < maxThreadsPerBlock);
    std::cout << "threadsPerBlock (used/max): " << threadsPerBlock << "/" << maxThreadsPerBlock << std::endl;

    // Check memory requirmenets
    long maxSharedMemoryPerBlock = prop.sharedMemPerBlock;
    long sharedMemoryPerBlock = 2*threadsPerBlock*sizeof(float);
    assert(sharedMemoryPerBlock <=  maxSharedMemoryPerBlock);
    std::cout << "Shared memory per block (used/max): " << sharedMemoryPerBlock << "/" << maxSharedMemoryPerBlock << std::endl;
    
    // Determine number of blocks to use
    int limitBlocksDueToSMem = maxSharedMemoryPerBlock / sharedMemoryPerBlock;
    int limitBlocksDueToWarps = threadsPerBlock / warpSize;
    int blocksPerMP = std::min(limitBlocksDueToSMem, limitBlocksDueToWarps);

    std::cout << "Active thread blocks per MP: " << blocksPerMP << std::endl;
    int blocks = blocksPerMP*prop.multiProcessorCount;
    std::cout << "Num blocks: " << blocks << std::endl;
    std::cout << "Total shared memory (used/max): " << sharedMemoryPerBlock*blocks << "/" << maxSharedMemoryPerBlock << std::endl;

    int nhistbins = threadsPerBlock;
    
    assert(nhistbins >= nbins+2);

    float dsum[nhistbins];
    float wsum[nhistbins];

    std::vector<double> tempxi(nbins,0);
    std::vector<double> counts(nbins,0);

    // allocate memory on the GPU for the file's data
    DataStruct *dev_b1, *dev_b2;
    float *dev_dsum;
    float *dev_wsum;

    HANDLE_ERROR( hipMalloc( (void**)&dev_b1, chunksize * sizeof(DataStruct) ) ); 
    HANDLE_ERROR( hipMalloc( (void**)&dev_b2, chunksize * sizeof(DataStruct) ) ); 
    HANDLE_ERROR( hipMalloc( (void**)&dev_dsum, nhistbins * sizeof( float ) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_wsum, nhistbins * sizeof( float ) ) );

    hipEvent_t start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );

    long totalcounts = 0;
    double totalElapsedTime = 0;
    for(int ichunk = 0; ichunk < nchunks; ++ichunk) {
        
        HANDLE_ERROR( hipEventRecord( start, 0 ) );

        for(int jchunk = 0; jchunk <= ichunk; ++jchunk) {

            //std::cout << "Starting chunk (" << ichunk << "," << jchunk << ")..." << std::endl;

            HANDLE_ERROR( hipMemcpy( dev_b1, &data[ichunk*chunksize], 
                chunksize * sizeof(DataStruct), hipMemcpyHostToDevice ) );
            HANDLE_ERROR( hipMemcpy( dev_b2, &data[jchunk*chunksize], 
                chunksize * sizeof(DataStruct), hipMemcpyHostToDevice ) );
            HANDLE_ERROR( hipMemset( dev_dsum, 0, nhistbins * sizeof( float ) ) );
            HANDLE_ERROR( hipMemset( dev_wsum, 0, nhistbins * sizeof( float ) ) );
        
            histo_kernel<<<blocks, threadsPerBlock, sharedMemoryPerBlock>>>(dev_b1, dev_b2, 
                chunksize, dev_dsum, dev_wsum, min, max, nbins, nhistbins, ichunk == jchunk);

            HANDLE_ERROR( hipMemcpy( dsum, dev_dsum, nhistbins * sizeof( float ), hipMemcpyDeviceToHost ) );
            HANDLE_ERROR( hipMemcpy( wsum, dev_wsum, nhistbins * sizeof( float ), hipMemcpyDeviceToHost ) );

            long chunkcounts = 0;
            // Save results from chunk
            //std::cout << wsum[0] << " " << wsum[nbins+1] << std::endl;
            for(int i = 0; i < nhistbins; ++i) {
                chunkcounts += wsum[i];
                if (i <= nbins && i > 0) {
                    //std::cout << i-1 << " " << dsum[i] << std::endl;
                    tempxi[i-1] += dsum[i];
                    counts[i-1] += wsum[i];
                }
            }
            totalcounts += chunkcounts;

            //std::cout << "Chunk (" << ichunk << "," << jchunk << ") counts: " << chunkcounts << std::endl;

            hipDeviceSynchronize();

        }
        // get stop time, and display the timing results
        HANDLE_ERROR( hipEventRecord( stop, 0 ) ); 
        HANDLE_ERROR( hipEventSynchronize( stop ) ); 
        float elapsedTime;
        HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
        totalElapsedTime += elapsedTime;
        printf( "Time to generate (%d):  %3.1f ms\n", ichunk, elapsedTime );
    }

    std::cout << "Total elapsed time: " << totalElapsedTime << " ms" << std::endl;

    long usedcounts = 0;
    for(int i = 0; i < nbins; ++i) {
        usedcounts += counts[i];
        if(counts[i] > 0) tempxi[i] /= counts[i];
    }

    tempxi.swap(xi);

    std::cout << "used " << usedcounts << " of " << totalcounts << " pairs." << std::endl;

    // Free host and device memory
    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );
    hipFree( dev_dsum ); 
    hipFree( dev_wsum );
    hipFree( dev_b1 ); 
    hipFree( dev_b2 ); 
    free(data);

}

int main(int argc, char **argv) {

    // Configure command-line option processing
    std::string infile,outfile,axis1,axis2;
    long chunksize;
    po::options_description cli("Correlation function estimator");
    cli.add_options()
        ("help,h", "Prints this info and exits.")
        ("verbose", "Prints additional information.")
        ("input,i", po::value<std::string>(&infile)->default_value(""),
            "Filename to read field samples from")
        ("output,o", po::value<std::string>(&outfile)->default_value("xi.dat"),
            "Filename to write correlation function to")
        ("axis1", po::value<std::string>(&axis1)->default_value("[0:200]*50"),
            "Axis-1 binning")
        ("axis2", po::value<std::string>(&axis2)->default_value("[0:200]*50"),
            "Axis-2 binning")
        ("rmu", "Use (r,mu) binning instead of (rP,rT) binning")
        ("chunksize", po::value<long>(&chunksize)->default_value(4096),
            "Number of chunks to split the dataset into.")
        ;

    // do the command line parsing now
    po::variables_map vm;
    try {
        po::store(po::parse_command_line(argc, argv, cli), vm);
        po::notify(vm);
    }
    catch(std::exception const &e) {
        std::cerr << "Unable to parse command line options: " << e.what() << std::endl;
        return -1;
    }
    if(vm.count("help")) {
        std::cout << cli << std::endl;
        return 1;
    }
    bool verbose(vm.count("verbose")),rmu(vm.count("rmu"));

    // Read the input file
    if(0 == infile.length()) {
        std::cerr << "Missing infile parameter." << std::endl;
        return -2;
    }
    std::vector<std::vector<double> > columns(5);
    try {
        std::ifstream in(infile.c_str());
        lk::readVectors(in,columns);
        in.close();
    }
    catch(std::exception const &e) {
        std::cerr << "Error while reading " << infile << ": " << e.what() << std::endl;
        return -3;
    }
    if(verbose) {
        std::cout << "Read " << columns[0].size() << " rows from " << infile
            << std::endl;
    }

    // Generate the correlation function grid and run the estimator
    std::vector<double> xi;
    try {
        lk::AbsBinningCPtr bins1 = lk::createBinning(axis1), bins2 = lk::createBinning(axis2);
        double x1min(bins1->getBinLowEdge(0)), x1max(bins1->getBinHighEdge(bins1->getNBins()-1));
        double x2min(bins2->getBinLowEdge(0)), x2max(bins2->getBinHighEdge(bins2->getNBins()-1));
        lk::BinnedGrid grid(bins1,bins2);
        int x1nbins = bins1->getNBins();

        bruteGPU(columns,x1min,x1max,x1nbins,xi,chunksize);
    }
    catch(std::exception const &e) {
        std::cerr << "Error while running the estimator: " << e.what() << std::endl;
    }

    // Save the estimator results
    try {
        std::ofstream out(outfile.c_str());
        for(int index = 0; index < xi.size(); ++index) {
            out << index << ' ' << xi[index] << std::endl;
        }
        out.close();
    }
    catch(std::exception const &e) {
        std::cerr << "Error while saving results: " << e.what() << std::endl;
    }

    return 0;
}
